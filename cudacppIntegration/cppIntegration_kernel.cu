/*
 * Copyright 1993-2009 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */

/* Example of integrating CUDA functions into an existing 
 * application / framework.
 * Device code.
 */

#ifndef _CPP_INTEGRATION_KERNEL_H_
#define _CPP_INTEGRATION_KERNEL_H_

///////////////////////////////////////////////////////////////////////////////
//! Simple test kernel for device functionality
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernel( int* g_data )
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid 
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be 
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp 
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid] = ((((data <<  0) >> 24) - 10) << 24)
                | ((((data <<  8) >> 24) - 10) << 16)
                | ((((data << 16) >> 24) - 10) <<  8)
                | ((((data << 24) >> 24) - 10) <<  0);
}

///////////////////////////////////////////////////////////////////////////////
//! Demonstration that int2 data can be used in the cpp code
//! @param g_odata  memory to process (in and out)
///////////////////////////////////////////////////////////////////////////////
__global__ void
kernel2( int2* g_data )
{
    // write data to global memory
    const unsigned int tid = threadIdx.x;
    int2 data = g_data[tid];

    // use integer arithmetic to process all four bytes with one thread
    // this serializes the execution, but is the simplest solutions to avoid 
    // bank conflicts for this very low number of threads
    // in general it is more efficient to process each byte by a separate thread,
    // to avoid bank conflicts the access pattern should be 
    // g_data[4 * wtid + wid], where wtid is the thread id within the half warp 
    // and wid is the warp id
    // see also the programming guide for a more in depth discussion.
    g_data[tid].x = data.x - data.y;
}

__global__ void
simpleAdd( float* A, float* B, float* C )
{
	int i = threadIdx.x;
		C[i]=A[i]+B[i];
}

#endif // #ifndef _CPP_INTEGRATION_KERNEL_H_
